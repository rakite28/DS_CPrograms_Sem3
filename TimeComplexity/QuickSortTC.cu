

#include <hip/hip_runtime.h>
#include<stdio.h>

#include<time.h>


__global__
void quickSort(int *a ,int first , int last)
{
    int i,j,pivot,temp;
    if(first<last)
    {
        pivot=first;
        i=first;
        j=last;
        while(i<j)
        {
            while(a[i]<=a[pivot]&&i<last)
            i++;
            while(a[j]>a[pivot])
            j--;
            if(i<j)
            {
                temp=a[i];
                a[i]=a[j];
                a[j]=temp;
            }
        }
        temp=a[pivot];
        a[pivot]=a[j];
        a[j]=temp;
        quickSort<<<1,1>>>(a,first,j-1);
        quickSort<<<1,1>>>(a,j+1,last);
    }
}

int main(void)
{
    int i,*a,n,l,r;
    n=1<<10;
    hipMallocManaged(&a,n*sizeof(int));
    clock_t start_t, end_t;
    double total_t;
    srand(time(0));
    FILE *fpt;
    fpt = fopen("QuickSortTC.csv", "w+");
    fprintf(fpt,"n,time_taken\n");
for(i=1;i<=n;i++)
{   
    for(int j=0;j<n;j++)
    {
        a[j]=(float)rand();
    }
    start_t = clock();
    l=0;
    r=n-1;
    quickSort<<<1,1>>>(a,l,r);
    hipDeviceSynchronize();
    // for(i=0;i<n;i++)
    // {
    //     printf("%d\n",a[i]);
    // }
    end_t = clock();
    total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    printf("Total time taken by CPU: %f\n", total_t  );
    

   fprintf(fpt,"%d, %f\n",i,total_t);
}
hipFree(a);
fclose(fpt);
return 0;
}